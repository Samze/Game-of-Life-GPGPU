#include "hip/hip_runtime.h"
#include "cellularautomata_kernal.h"

#include <hip/hip_vector_types.h>
#include <cstdio>

//extern "C" bool
extern "C" float CUDATimeStep(int* pFlatGrid, int DIM) {

	int *dev_pFlatGrid; //Pointers to device allocated memory
	int *dev_DIM;

	hipEvent_t start,stop; //Events for timings

	//START: Record duration of GPGPU processing
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	size_t noCells = DIM * DIM * sizeof(int);
	//Might need to flatten the 2d array ormaybe try "int2" type
	
	//Allocate suitable size memory on device
	hipMalloc((void**) &dev_pFlatGrid, noCells);
	hipMalloc((void**) &dev_DIM, sizeof(int));

	//Make our 2D grid of blocks & threads (DIM/No of threads)
	dim3 blocks (DIM/10, DIM/10);
	dim3 threads(10,10);
	
	//Copy our memory from Host to Device
	hipMemcpy(dev_pFlatGrid, pFlatGrid, noCells,
		hipMemcpyHostToDevice);
	hipMemcpy(dev_DIM, &DIM, sizeof(int),
		hipMemcpyHostToDevice);

	//Probably really bad way to do this....look into const mem
	//hipMemcpy(dev_DIM, pDim, sizeof(int),
	//	hipMemcpyHostToDevice);

	kernal<<<blocks,threads>>>(dev_pFlatGrid, dev_DIM);

	//Copy back to host
	hipMemcpy(pFlatGrid, dev_pFlatGrid, noCells,
		hipMemcpyDeviceToHost);

	//STOP : processing done
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//fix up states - normalize
	for (int i = 0; i < DIM; ++i) {
		for (int j = 0; j < DIM; ++j) {
				pFlatGrid[i * DIM +j] = pFlatGrid[i * DIM +j] >> 1;
		}
	}

	//Free memory on Device
	hipFree(dev_pFlatGrid);
	hipFree(dev_DIM);

	return elapsedTime;
}
