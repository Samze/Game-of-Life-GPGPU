#include "hip/hip_runtime.h"
#include "cellularautomata_kernal.h"

__global__ void kernal(int* g_data, int* DIM) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

	//calculates number of live neighbour cells
	int neighLiveCells = liveCellsInNeigh(g_data, x * *DIM, y, *DIM);	

	//set new cell state.
	g_data[(x * *DIM) + y] = applyRules(g_data[x * *DIM + y],neighLiveCells);

}


//probably a much better way to figure out the moores neighbourhood
__device__ int liveCellsInNeigh(int* g_data, int x, int y, int xDIM) {
	//get neighbours for cell x,y

	int numlivecells = 0;

	// [-1,-1]
	if (x != 0 && y != 0)
		if (g_data[x - (xDIM * 1) + y - 1] & 1 == 1)
			++numlivecells;

	// [0,-1]
	if ( y != 0)
		if (g_data[x + y - 1] & 1 == 1)
			++numlivecells;

	// [1,-1]
	if (x != xDIM - 1 && y != 0 )
		if (g_data[x + (xDIM * 1) + y - 1] & 1 == 1)
			++numlivecells;
	
	// [-1,0]
	if (x != 0)
		if (g_data[x - (xDIM * 1) + y] & 1 == 1)
			++numlivecells;	
	
	// [1,0]
	if (x != xDIM - 1)
		if (g_data[x + (xDIM * 1) + y] & 1 == 1)
			++numlivecells;

	// [-1,1]
	if (x != 0 && y != xDIM -1 )
		if (g_data[x - (xDIM * 1) + y + 1] & 1 == 1)
			++numlivecells;

	// [0,1]
	if (y != xDIM -1 )
		if (g_data[x + y + 1] & 1 == 1)
			++numlivecells;

	// [1,1]
	if (x != xDIM -1 && y != xDIM -1 )
		if (g_data[x + (xDIM * 1) + y + 1] & 1 == 1)
			++numlivecells;

	return numlivecells;
}

__device__ int applyRules(int state, int liveCells) {    
	//Any live cell with fewer than two live neighbours dies, as if caused by under-population.
	if (state && liveCells < 2)  return state;
	
	//Any live cell with two or three live neighbours lives on to the next generation.
	if (state && liveCells > 1 && liveCells < 4) return state | 2;
	
	//Any live cell with more than three live neighbours dies, as if by overcrowding.
	if (state && liveCells > 3) return state;

	//Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
	if (!state && liveCells == 3) return state | 2;

	//default
	return 0;
}